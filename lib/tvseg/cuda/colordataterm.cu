#include "hip/hip_runtime.h"

#include "colordataterm.h"
#include "errorhandling.h"
#include "config.h"

namespace tvseg {
namespace cuda {


__global__ void kernel_colorDataterm(
        float* dataterm,
        const float *image,
        const float *colors,
        const uint pitchDataterm,
        const uint pitchImage,
        const Dim3 dim)
{

    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= dim.width || y >= dim.height)
        return;

    const uint datatermImageSize = dim.height * pitchDataterm;

    for (uint i = 0; i < dim.labels; ++i) {
        const float r = image[3*x + y*pitchImage    ] - colors[3*i    ];
        const float g = image[3*x + y*pitchImage + 1] - colors[3*i + 1];
        const float b = image[3*x + y*pitchImage + 2] - colors[3*i + 2];

        dataterm[x + y*pitchDataterm + i*datatermImageSize] = sqrtf(r*r + g*g + b*b);
    }
}

void computeColorDataterm(
        float* dataterm,
        const float *image,
        const float *colors,
        const Dim3 &dim)
{
    // Prepare grid size
    dim3 dimBlock(BLOCKDIMX, BLOCKDIMY);
    dim3 dimGrid;
    size_t pitchDataterm;
    size_t pitchImage;
    size_t pitchColors;
    dimGrid.x = (dim.width % dimBlock.x) ? (dim.width/dimBlock.x + 1) : (dim.width/dimBlock.x);
    dimGrid.y = (dim.height % dimBlock.y) ? (dim.height/dimBlock.y + 1) : (dim.height/dimBlock.y);

    // allocate memory on the GPU and copy data or set initial value
    float *gpu_image = NULL;
    float *gpu_dataterm = NULL;
    float *gpu_colors = NULL;

    CUDA_CHECK(hipMallocPitch((void**)&gpu_dataterm, &pitchDataterm, dim.width * sizeof(float), dim.height * dim.labels));
    CUDA_CHECK(hipMemset2DAsync(gpu_dataterm, pitchDataterm, 0, dim.width * sizeof(float), dim.height * dim.labels));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_image, &pitchImage, 3*dim.width * sizeof(float), dim.height));
    CUDA_CHECK(hipMemcpy2DAsync(gpu_image, pitchImage, image, 3*dim.width * sizeof(float), 3*dim.width * sizeof(float), dim.height, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_colors, &pitchColors, 3*dim.labels * sizeof(float), 1));
    CUDA_CHECK(hipMemcpy2DAsync(gpu_colors, pitchColors, colors, 3*dim.labels * sizeof(float), 3*dim.labels * sizeof(float), 1, hipMemcpyHostToDevice));

    // compute dataterm on GPU
    kernel_colorDataterm<<< dimGrid, dimBlock >>>(gpu_dataterm, gpu_image, gpu_colors,
                                                  pitchDataterm/sizeof(float), pitchImage/sizeof(float),
                                                  dim);
    CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

    // copy result back to CPU
    CUDA_CHECK(hipMemcpy2D((void*)dataterm, dim.width * sizeof(float), gpu_dataterm, pitchDataterm, dim.width * sizeof(float), dim.height * dim.labels, hipMemcpyDeviceToHost));

    // delete all data on GPU
    CUDA_CHECK(hipFree(gpu_image));
    CUDA_CHECK(hipFree(gpu_dataterm));
    CUDA_CHECK(hipFree(gpu_colors));
}


}
}
